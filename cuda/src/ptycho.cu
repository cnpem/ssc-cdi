#include "hip/hip_runtime.h"
#include <cmath>
#include <common/types.hpp>
#include <common/logger.hpp>
#include <cmath>
#include <cstddef>

#include "complex.hpp"
#include "ptycho.hpp"

__global__ void KProjectPhiToProbe(const GArray<complex> probe, complex* probe_acc, float* probe_div,
        const GArray<complex> object, const GArray<complex> exitwave, const GArray<ROI> rois,
        bool bFTNorm, bool bIsGrad) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= probe.shape.x) return;

    complex pacc = complex(0);
    float pdiv = 0;

    for (size_t roi = 0; roi < rois.shape.z; roi++)
        for (int p = 0; p < rois.shape.x; p++)  // for each flyscan point
        {
            int objposx = idx + (int)rois(roi, 0, p).x;
            int objposy = idy + (int)rois(roi, 0, p).y;

            complex obj = object(objposy, objposx);
            complex ew = exitwave((roi * rois.shape.x + p) * probe.shape.z + blockIdx.z, idy, idx);

            pacc += ew * obj.conj();
            pdiv += obj.abs2();
        }

    size_t index = blockIdx.z * probe.shape.x * probe.shape.y + idy * probe.shape.x + idx;

    if (bFTNorm) pacc /= (float)(probe.shape.x * probe.shape.y);
    if (!bIsGrad) pacc -= probe[index] * pdiv;

    probe_acc[index] += pacc;
    probe_div[index] += pdiv;
}

// the kernel code is replicated for complex16, for some reason cuda was not playing well with explicit instantiation on gpu kernels
__global__ void KProjectPhiToProbe(const GArray<complex> probe, complex* probe_acc, float* probe_div,
        const GArray<complex> object, const GArray<complex16> exitwave, const GArray<ROI> rois,
        bool bFTNorm, bool bIsGrad) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= probe.shape.x) return;

    complex pacc = complex(0);
    float pdiv = 0;

    for (size_t roi = 0; roi < rois.shape.z; roi++)
        for (int p = 0; p < rois.shape.x; p++)  // for each flyscan point
        {
            int objposx = idx + (int)rois(roi, 0, p).x;
            int objposy = idy + (int)rois(roi, 0, p).y;

            complex obj = object(objposy, objposx);
            complex ew = complex(exitwave((roi * rois.shape.x + p) * probe.shape.z + blockIdx.z, idy, idx));

            pacc += ew * obj.conj();
            pdiv += obj.abs2();
        }

    size_t index = blockIdx.z * probe.shape.x * probe.shape.y + idy * probe.shape.x + idx;

    if (bFTNorm) pacc /= (float)(probe.shape.x * probe.shape.y);
    if (!bIsGrad) pacc -= probe[index] * pdiv;

    probe_acc[index] += pacc;
    probe_div[index] += pdiv;
}

template <typename dtype>
void ProjectPhiToProbe(POptAlgorithm& pt, int section, const MImage<dtype>& Phi, bool bNormalizeFFT, bool bIsGradPm) {
    dim3 blk = pt.probe->ShapeBlock();
    dim3 thr = pt.probe->ShapeThread();

    for (int g = 0; g < pt.gpus.size(); g++) {
        SetDevice(pt.gpus, g);

        KProjectPhiToProbe<<<blk, thr>>>(pt.probe->arrays[g][0], pt.probe_acc->Ptr(g), pt.probe_div->Ptr(g),
                pt.object->arrays[g][0], Phi.arrays[g][0], pt.rois[section]->arrays[g][0], bNormalizeFFT, bIsGradPm);
    }
}

template void ProjectPhiToProbe<complex>(POptAlgorithm& pt, int section, const cMImage& Phi, bool bNormalizeFFT, bool bIsGradPm);

template void ProjectPhiToProbe<complex16>(POptAlgorithm& pt, int section, const hcMImage& Phi, bool bNormalizeFFT, bool bIsGradPm);

extern "C" {
    void EnablePeerToPeer(const std::vector<int>& gpus);
    void DisablePeerToPeer(const std::vector<int>& gpus);


    __global__ void k_project_reciprocal_space(GArray<complex> exitwave,
            const GArray<float> difpads, float* rfactors,
            size_t upsample, size_t nummodes,
            int geometricsteps, bool bIsGrad) {
        __shared__ float sh_rfactor[64];

        if (threadIdx.x < 64) sh_rfactor[threadIdx.x] = 0;

        __syncthreads();


        const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
        const size_t idy = blockIdx.y;
        const size_t idz = blockIdx.z;

        if (idx >= difpads.shape.x) return;

        const float difpad = difpads(idz, idy, idx);
        const float sqrt_difpad = sqrtf(difpad);

        float exit_wave_factor = 1.0f;
        float exit_wave_addend = 0.0f;
        if (difpad >= 0) {
            float wabs2 = 0.0f;

            for (int m = 0; m < nummodes; m++)
                for (int f = 0; f < geometricsteps; f++)
                    for (int v = 0; v < upsample; v++)
                        for (int u = 0; u < upsample; u++)
                            wabs2 += exitwave(geometricsteps * idz * nummodes + nummodes * f + m,
                                    v + idy * upsample,
                                    u + idx * upsample).abs2();

            wabs2 = sqrtf(wabs2 / geometricsteps) / upsample;

            const float hexptaulambda2 = 1.0;

            atomicAdd(sh_rfactor + threadIdx.x % 64, sq(sqrt_difpad - wabs2 * hexptaulambda2));

            if (wabs2 > 0.0f) {
                exit_wave_factor = (sqrt_difpad / wabs2 - 1);
                if (!bIsGrad) exit_wave_factor += 1.0f;
                exit_wave_addend = 0.0f;
            } else {
                exit_wave_addend = sqrt_difpad;
                exit_wave_factor = 0.0f;
            }
        } else if (bIsGrad) {
            exit_wave_factor = 0.0f;
            exit_wave_addend = 0.0f;
        }

        for (int m = 0; m < nummodes; m++)
            for (int f = 0; f < geometricsteps; f++)
                for (int v = 0; v < upsample; v++)
                    for (int u = 0; u < upsample; u++) {
                        complex ew = exitwave(geometricsteps * idz * nummodes + nummodes * f + m,
                                v + idy * upsample,
                                u + idx * upsample);

                        ew = ew * exit_wave_factor + exit_wave_addend; //possibly has to deal with nan or inf?
                        exitwave(geometricsteps * idz * nummodes + nummodes * f + m,
                                v + idy * upsample,
                                u + idx * upsample) = ew;
                    }

        __syncthreads();

        Reduction::KSharedReduce(sh_rfactor, 64);
        if (threadIdx.x == 0) atomicAdd(rfactors + blockIdx.y, sh_rfactor[0]);
    }
}


void IndexRois(ROI* rois, int numrois) {
    for(int r=0; r<numrois; r++) rois[r].I0 = (float)r;
}


void project_reciprocal_space(POptAlgorithm &pt, rImage* difpad, int g, bool bIsGradPm) {

    SetDevice(pt.gpus, g);

    complex* ewave = pt.exitwave->Ptr(g);

    int upsample = pt.exitwave->sizex / difpad->sizex;

    pt.propagator[g]->Propagate(ewave, ewave, pt.exitwave->Shape(), 1);

    pt.exitwave->arrays[g]->FFTShift2();

    k_project_reciprocal_space<<<difpad->ShapeBlock(), difpad->ShapeThread()>>>(pt.exitwave->arrays[g][0], *difpad, pt.rfactors->Ptr(g), upsample,
            pt.probe->sizez, pt.geometricsteps, bIsGradPm);


    pt.exitwave->arrays[g]->FFTShift2();
    pt.propagator[g]->Propagate(ewave, ewave, pt.exitwave->Shape(), -1);

}


void ApplyProbeUpdate(POptAlgorithm& pt, cImage& velocity, float stepsize, float momentum, float epsilon) {

    if (momentum < 0 | stepsize < 0) return;



    SetDevice(pt.gpus, 0);


    pt.probe->WeightedLerpSync(*(pt.probe_acc), *(pt.probe_div), stepsize, momentum, velocity, epsilon);

    if (pt.probesupport != nullptr) {
        dim3 shape = dim3(pt.probe->sizex, pt.probe->sizey, pt.probe->sizez);
        complex* pointer = pt.probe->arrays[0]->gpuptr;
        SetDevice(pt.gpus, 0);

        if (pt.probef1 != 0) pt.probepropagator->Propagate(pointer, pointer, shape, +pt.probef1);

        pt.probe->arrays[0][0] *= pt.probesupport->arrays[0][0];

        if (pt.probef1 != 0) pt.probepropagator->Propagate(pointer, pointer, shape, -pt.probef1);

        pt.probe->BroadcastSync();
    }
}

void DestroyPOptAlgorithm(POptAlgorithm*& ptycho_ref) {
    POptAlgorithm& ptycho = *ptycho_ref;
    ssc_debug("Dealloc POpt.");
    if (ptycho.object_div) delete ptycho.object_div;
    ptycho.object_div = nullptr;
    if (ptycho.object_acc) delete ptycho.object_acc;
    ptycho.object_acc = nullptr;
    if (ptycho.probe_div) delete ptycho.probe_div;
    ptycho.probe_div = nullptr;
    if (ptycho.probe_acc) delete ptycho.probe_acc;
    ptycho.probe_acc = nullptr;

    ssc_debug("Deallocating base algorithm.");
    for (int g = 0; g < ptycho.gpus.size(); g++) {
        ssc_debug(format("Dealloc propagator: {}", g));
        SetDevice(ptycho.gpus, g);
        delete ptycho.propagator[g];
        ptycho.propagator[g] = nullptr;
    }

    ssc_debug("Probe D2H");
    ptycho.probe->CopyTo(ptycho.cpuprobe);
    ssc_debug("Object D2H");
    ptycho.object->CopyTo(ptycho.cpuobject);

    ssc_debug("Dealloc probe.");
    delete ptycho.probe;
    ssc_debug("Dealloc object.");
    delete ptycho.object;
    ssc_debug("Dealloc exitwave.");
    delete ptycho.exitwave;

    ssc_debug("Dealloc supports.");
    if (ptycho.objectsupport != nullptr) delete ptycho.objectsupport;
    if (ptycho.probesupport != nullptr) delete ptycho.probesupport;

    ssc_debug("Dealloc rfactors.");
    delete ptycho.rfactors;

    ssc_debug("Dealloc rois.");
    for (auto* roi : ptycho.rois) delete roi;

    ssc_debug("Done.");

    SetDevice(ptycho.gpus, 0);
    delete ptycho.probepropagator;

    ptycho_ref = nullptr;
}

POptAlgorithm* CreatePOptAlgorithm(float* _difpads, const dim3& difshape, complex* _probe, const dim3& probeshape,
            complex* _object, const dim3& objshape, ROI* _rois, int numrois, int batchsize,
            float* _rfact, const std::vector<int>& gpus, float* _objectsupport, float* _probesupport,
            int numobjsupp, int geometricsteps, float probef1,
            float step_obj, float step_probe,
            float reg_obj, float reg_probe) {

    POptAlgorithm* ptycho = new POptAlgorithm;
     ptycho->gpus = gpus;

      ssc_debug("Initializing algorithm.");
            ssc_debug("Enabling P2P");

            ptycho->probef1 = probef1;
            EnablePeerToPeer(ptycho->gpus);

            ptycho->objreg = reg_obj;
            ptycho->probereg = reg_probe;
            ptycho->objstep = step_obj;
            ptycho->probestep = step_probe;

            ptycho->difpadshape.x = difshape.x;
            ptycho->difpadshape.y = difshape.y;
            ptycho->difpadshape.z = difshape.z;

            const int ngpus = gpus.size();
            ptycho->geometricsteps = geometricsteps;
            if (batchsize > 0) {
                ptycho->singlebatchsize = batchsize;
                ptycho->multibatchsize = batchsize * ngpus;

                batchsize *= ngpus;
            } else {
                ptycho->singlebatchsize = (numrois + ngpus - 1) / ngpus;
                batchsize = ptycho->multibatchsize = ptycho->singlebatchsize * ngpus;
            }
            ssc_debug(format("Batches: {} {}", ptycho->singlebatchsize, ptycho->multibatchsize));

            ptycho->total_num_rois = numrois;

            ptycho->cpudifpads = _difpads;
            ptycho->cpuprobe = _probe;
            ptycho->cpuobject = _object;
            ptycho->cpurois = _rois;
            ptycho->cpurfact = _rfact;

            ssc_debug("Alloc probe.");
            ptycho->probe = new cMImage(_probe, probeshape, true, gpus);
            ssc_debug("Alloc obj");
                ptycho->object = new cMImage(_object, objshape, true, gpus);
            ssc_debug("Alloc EW");
            ptycho->exitwave = new cMImage(probeshape.x, probeshape.y,
                    ptycho->singlebatchsize * probeshape.z * geometricsteps, true, gpus);

            ssc_debug("Alloc Supports");

            if (numobjsupp > 0 && _objectsupport != nullptr) {
                ptycho->objectsupport = new rMImage(_objectsupport, dim3(objshape.x, objshape.y, numobjsupp), true, ptycho->gpus);
                ptycho->SupportSizes = std::vector<float>();
                for (int i = 0; i < numobjsupp; i++) {
                    float s = 0;
                    for (int j = 0; j < objshape.x * objshape.y; j++) s += _objectsupport[j + i * objshape.x * objshape.y];
                    ptycho->SupportSizes.push_back(s);
                }
            } else
                ptycho->objectsupport = nullptr;

            if (_probesupport != nullptr)
                ptycho->probesupport = new rMImage(_probesupport, dim3(probeshape.x, probeshape.y, probeshape.z), true, ptycho->gpus);
            else
                ptycho->probesupport = nullptr;

            ssc_debug("Alloc RF");
            ptycho->rfactors = new rMImage(difshape.y, 1, 1, true, ptycho->gpus);
            ptycho->rfactors->SetGPUToZero();

            SetDevice(ptycho->gpus, 0);
            ptycho->roibatch_offset = std::vector<int>();

            for (size_t n = 0; n < numrois; n += batchsize) {
                ssc_debug(format("Creating DPGroup at: {} of {} at step {}" , n, numrois, batchsize));
                if (numrois - n < batchsize)  // last batch
                {
                    ptycho->rois.push_back(new RoiArray(_rois + n * geometricsteps, geometricsteps, 1, numrois - n, false, gpus));
                } else {
                    ptycho->rois.push_back(new RoiArray(_rois + n * geometricsteps, geometricsteps, 1, batchsize, false, gpus));
                }
                ptycho->roibatch_offset.push_back(n / ngpus);
            }

            for (int g = 0; g < gpus.size(); g++) {
                SetDevice(gpus, g);
                ssc_debug(format("Creating propagator: {}", g));
                ptycho->propagator[g] = new Fraunhoffer();
            }

            ssc_debug("Computing I0");
            SetDevice(gpus, 0);
            ptycho->I0 = ptycho->probe->arrays[0]->Norm2();
            ptycho->probepropagator = new ASM();

            ptycho->object_div = new rMImage(ptycho->object->Shape(), true, gpus);
            ptycho->object_acc = new cMImage(ptycho->object->Shape(), true, gpus);

            ptycho->probe_div = new rMImage(ptycho->probe->Shape(), true, gpus);
            ptycho->probe_acc = new cMImage(ptycho->probe->Shape(), true, gpus);

            return ptycho;
}
