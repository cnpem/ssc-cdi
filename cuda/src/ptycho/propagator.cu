#include "hip/hip_runtime.h"
#include "logger.hpp"
#include <hipfft/hipfft.h>
#include <hip/driver_types.h>

#include <propagator.hpp>

extern "C" {
void Fraunhoffer::Append(const dim3& dim, hipStream_t stream) {
    std::vector<dim3>& dims = st_dims[stream];
    std::vector<hipfftHandle>& plans = st_plans[stream];

    sscDebug(format("Creating new plan: {} {} {}", dim.x, dim.y, dim.z));
    if (workarea == nullptr || dim.x * dim.y * dim.z > workarea->size) {
        sscDebug(format("Reallocating plan memory to size: {} {} {}", dim.x,
                        dim.y, dim.z));
        if (workarea) delete workarea;
        workarea = new cImage(dim.x, dim.y, dim.z);

        for (auto plan : plans) hipfftSetWorkArea(plan, workarea->gpuptr);
    }

    int i = dims.size();
    dims.push_back(dim);

    int n[] = {(int)dim.x, (int)dim.y};

    hipfftHandle newplan;

    size_t worksize;
    sscCufftCheck(hipfftCreate(&newplan));
    sscCufftCheck(hipfftSetAutoAllocation(newplan, 0));
    sscCufftCheck(hipfftMakePlanMany(newplan, 2, n, nullptr, 0, 0, nullptr, 0, 0,
                                    HIPFFT_C2C, (int)dim.z, &worksize));
    sscCufftCheck(hipfftSetWorkArea(newplan, workarea->gpuptr));
    sscCufftCheck(hipfftSetStream(newplan, stream));

    plans.push_back(newplan);

    sscAssert(worksize <= 8 * workarea->size, "CuFFT being hungry!");
    sscDebug("Done.");
}

bool dim3EQ(const dim3& d1, const dim3& d2) {
    return d1.x == d2.x && d1.y == d2.y && d1.z == d2.z;
}

void Fraunhoffer::Propagate(complex* owave, complex* iwave, dim3 shape,
                            float amount, hipStream_t stream) {
    std::vector<dim3>& dims = st_dims[stream];
    std::vector<hipfftHandle>& plans = st_plans[stream];

    bool bPlanExists = false;
    auto dir = (amount > 0) ? HIPFFT_FORWARD : HIPFFT_BACKWARD;

    hipfftHandle plan;

    for (int i = 0; i < dims.size(); i++) {
        if (dim3EQ(shape, dims[i])) {
                bPlanExists = true;
                plan = plans[i];
        }
    }
    if (!bPlanExists) {
        Append(shape);
        plan = plans[plans.size() - 1];
    }
    sscCufftCheck(hipfftExecC2C(plan, iwave, owave, dir));
}

Fraunhoffer::~Fraunhoffer() {
    sscDebug("Deleting propagator.");
    for (auto& [key, plans] : st_plans) {
        for (auto plan : plans) {
            if (plan) hipfftDestroy(plan);
        }
    }
    if (workarea) delete workarea;
}

__global__ void KApplyASM(complex* wave, float fresnel_number, dim3 shape) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t idy = blockIdx.y;
    size_t idz = blockIdx.z;

    if (idx >= shape.x) return;

    float xx =
        float(int(idx + shape.x / 2) % int(shape.x)) / float(shape.x) - 0.5f;
    float yy =
        float(int(idy + shape.y / 2) % int(shape.y)) / float(shape.y) - 0.5f;

    wave[idz * shape.x * shape.y + idy * shape.x + idx] *=
        complex::exp1j(-float(M_PI) / fresnel_number * (xx * xx + yy * yy)) /
        float(shape.x * shape.y);
}

ASM::ASM(float wavelength, float pixelsize_m)
    : wavelength_m(wavelength_m), pixelsize_m(pixelsize_m) {}

void ASM::Propagate(complex* owave, complex* iwave, dim3 shape,
                    float distance_m, hipStream_t stream) {
    const float fresnel_number =
        (pixelsize_m * pixelsize_m) / wavelength_m / distance_m;
    Fraunhoffer::Propagate(owave, iwave, shape, 1, stream);
    const dim3 blk((shape.x + 127) / 128, shape.y, shape.z);
    const dim3 thr(128, 1, 1);
    KApplyASM<<<blk, thr, 0, stream>>>(owave, fresnel_number, shape);
    Fraunhoffer::Propagate(owave, owave, shape, -1, stream);
}
}
