#include "hip/hip_runtime.h"
#include <cstddef>

#include "ptycho.hpp"
#include <common/logger.hpp>
#include <common/types.hpp>
#include <common/utils.hpp>

extern "C"{

__global__ void KSideExitwave(GArray<complex> exitwave, const GArray<complex> probe, const GArray<complex> object, const GArray<Position> rois, int offx, int offy)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= probe.shape.x)
		return;

	int idy = blockIdx.y*blockDim.y + threadIdx.y;
	int idz = blockIdx.z;

	if(true)
	{
		int objposx = idx + (int)rois(idz,0,0).x + offx;
		int objposy = idy + (int)rois(idz,0,0).y + offy;

		const complex& obj = object(objposy, objposx);

		for(size_t m=0; m<probe.shape.z; m++) // for each incoherent mode
			exitwave(m + probe.shape.z*blockIdx.z,idy,idx) = obj * probe(m,idy,idx);
	}
}
__global__ void KComputeError(float* rfactors, const GArray<complex> exitwave, const GArray<float> difpads,
    const float* background, size_t nummodes)
{
    __shared__ float sh_rfactor[64];

    if(threadIdx.x<64)
        sh_rfactor[threadIdx.x] = 0;

    __syncthreads();

    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t idy = blockIdx.y;

    if(idx >= difpads.shape.x)
        return;

    bool bApplyBkg = background != nullptr;

    float difpad = difpads(blockIdx.z, idy, idx);

    if(difpad >= 0)
    {
        float wabs2 = 0.0f;
        if( bApplyBkg ) wabs2 = sq( background[idy*difpads.shape.x+idx] );

        for(int m=0; m<nummodes; m++)
            wabs2 += exitwave(blockIdx.z*nummodes + m, idy, idx).abs2();

        const int sigmask = (difpad < 0);
        atomicAdd(sh_rfactor + threadIdx.x%64, sigmask * sq(sqrtf(difpad)-sqrtf(wabs2)));
    }

    __syncthreads();

    Reduction::KSharedReduce(sh_rfactor,64);
    if(threadIdx.x==0)
        atomicAdd(rfactors + blockIdx.z, sh_rfactor[0]);
}

}

PosCorrection* CreatePosCorrection(float* difpads, const dim3& difshape, complex* probe, const dim3& probeshape,
                                   complex* object, const dim3& objshape, Position* rois, int numrois, int batchsize,
                                   float* rfact, const std::vector<int>& gpus, float* objsupp, float* probesupp,
                                   int numobjsupp,
                                   float wavelength_m, float pixelsize_m, float distance_m,
                                   float step_obj, float step_probe,
                                   float reg_obj, float reg_probe) {
    PosCorrection* poscorr = new PosCorrection;
    poscorr->errorcounter = new rMImage(5, 1, batchsize, true, gpus);
    poscorr->ptycho =
        CreatePOptAlgorithm(difpads, difshape, probe, probeshape,
                object, objshape, rois, numrois, batchsize, rfact,
                gpus, objsupp, probesupp, numobjsupp,
                wavelength_m, pixelsize_m, distance_m,
                step_obj, step_probe, reg_obj, reg_probe);
    return poscorr;
}

/**
* Positions are updated along with the probe.
* */
void PosCorrectionApplyProbeUpdate(cImage& velocity, float stepsize, float momentum, float epsilon);


void PosCorrectionProjectProbe(PosCorrection& poscorr, int section) {
    ProjectPhiToProbe(*poscorr.ptycho, section, *poscorr.ptycho->exitwave, true, poscorr.isGradPm);
}

void DestroyPosCorrection(PosCorrection*& poscorr) {
    delete poscorr->errorcounter;
    const size_t num_batches = poscorr->ptycho->positions.size();
    size_t global_idx = 0;
    for(int d = 0; d < num_batches; d++) {
        for(int g = 0; g<poscorr->ptycho->gpus.size(); g++) {
            for(size_t z = 0; z < poscorr->ptycho->positions[d][0][g].sizez; z++) {
                poscorr->ptycho->cpurois[global_idx].x = poscorr->ptycho->positions[d][0][g].cpuptr[z].x;
                poscorr->ptycho->cpurois[global_idx].y = poscorr->ptycho->positions[d][0][g].cpuptr[z].y;
                global_idx++;
            }
        }
    }
    DestroyPOptAlgorithm(poscorr->ptycho);
    poscorr = nullptr;
}

void PosCorrectionApplyProbeUpdate(PosCorrection& poscorr, cImage& velocity,
        float stepsize, float momentum, float epsilon) {
    POptAlgorithm& ptycho = *poscorr.ptycho;

    ApplyProbeUpdate(ptycho, velocity, stepsize, momentum, epsilon);

    float const offx[] = {0,1,-1,0,0};
    float const offy[] = {0,0,0,1,-1};

    const size_t batchsize = ptycho.positions[0]->arrays[0]->sizez;

    const dim3 difpadshape = ptycho.difpadshape;

    rMImage cur_difpad(difpadshape.x, difpadshape.y, ptycho.multibatchsize,
            false, ptycho.gpus, MemoryType::EAllocGPU);

    const size_t num_batches = ptycho_num_batches(ptycho);
    for(int d = 0; d<num_batches; d++) {
        const size_t difpad_batch_zsize = ptycho_cur_batch_zsize(ptycho, d);
        const size_t difpad_idx = d * ptycho_batch_size(ptycho);

        cur_difpad.Resize(difpadshape.x, difpadshape.y, difpad_batch_zsize);
        cur_difpad.LoadToGPU(ptycho.cpudifpads + difpad_idx * difpadshape.x * difpadshape.y);

        poscorr.errorcounter->SetGPUToZero();
        ptycho.positions[d]->LoadFromGPU();

        const size_t ngpus = ptycho_num_gpus(ptycho);
        for(int k = 0; k<5; k++)
            for(int g = 0; g < ngpus; g++) {
                const size_t difpadsizez = ptycho.positions[d][0][g].sizez;
                if(difpadsizez > 0) {
                    SetDevice(ptycho.gpus, g);
                    dim3 blk = ptycho.exitwave->ShapeBlock(); blk.z = difpadsizez;
                    dim3 thr = ptycho.exitwave->ShapeThread();

                    Image<Position>& ptr_roi = *ptycho.positions[d]->arrays[g];
                    KSideExitwave<<<blk,thr>>>(*ptycho.exitwave->arrays[g],
                            *ptycho.probe->arrays[g],
                            *ptycho.object->arrays[g],
                            ptr_roi, offx[k], offy[k]);
                    ptycho.propagator[g]->Propagate(ptycho.exitwave->arrays[g]->gpuptr,
                            ptycho.exitwave->arrays[g]->gpuptr,
                            ptycho.exitwave->arrays[g]->Shape(), 1);

                    KComputeError<<<blk,thr>>>(
                            poscorr.errorcounter->arrays[g]->gpuptr + batchsize*k,
                            *ptycho.exitwave->arrays[g], *cur_difpad.arrays[g],
                            nullptr,
                            ptycho.probe->sizez);
            }
        }

        poscorr.errorcounter->LoadFromGPU();
        poscorr.errorcounter->SyncDevices();

        for(int g = 0; g<ptycho.gpus.size(); g++) {
            const size_t batch_size = ptycho_cur_batch_gpu_zsize(ptycho, d, g);
            for(size_t z = 0; z < batch_size; z++) {
                float* error = poscorr.errorcounter->arrays[g]->cpuptr + z;
                float minerror = 1E35f;
                int minidx = 0;

                for(int k = 0; k<5; k++) if(error[batchsize*k] < minerror) {
                    minerror = error[batchsize*k];
                    minidx = k;
                }
                ptycho.positions[d][0][g].cpuptr[z].x = fminf(fmaxf(ptycho.positions[d][0][g].cpuptr[z].x+offx[minidx],1.1f),
                        ptycho.object->sizex - ptycho.probe->sizex-3);
                ptycho.positions[d][0][g].cpuptr[z].y = fminf(fmaxf(ptycho.positions[d][0][g].cpuptr[z].y+offy[minidx],1.1f),
                        ptycho.object->sizey - ptycho.probe->sizey-3);
            }
        }

        ptycho.positions[d]->LoadToGPU();
    }
    SyncDevices(ptycho.gpus);
}


void PosCorrectionRun(PosCorrection& poscorr, int iterations) {
  ssc_debug("Starting PosCorrectionRun.");

  POptAlgorithm& ptycho = *poscorr.ptycho;

  auto time0 = ssc_time();

  ptycho.object->Set(0);
  cImage objvelocity(ptycho.object->Shape());
  cImage probevelocity(ptycho.probe->Shape());
  objvelocity.SetGPUToZero();
  probevelocity.SetGPUToZero();


  const dim3 difpadshape = ptycho.difpadshape;

  for (int iter = 0; iter < iterations; iter++) {
    ssc_debug(format("Start PosCorr iteration: {}", iter));

    // std::cout << iter << std::endl;
    const bool bIterProbe = (ptycho.probemomentum >= 0);  // & (iter > iterations/20);
    ptycho.rfactors->SetGPUToZero();
    ptycho.object_acc->SetGPUToZero();
    ptycho.object_div->SetGPUToZero();
    ptycho.probe_acc->SetGPUToZero();
    ptycho.probe_div->SetGPUToZero();

    if (iter < 2) {
      objvelocity.SetGPUToZero();
      probevelocity.SetGPUToZero();
    }

    rMImage cur_difpad(difpadshape.x, difpadshape.y, ptycho.multibatchsize,
            false, ptycho.gpus, MemoryType::EAllocGPU);

    const size_t num_batches = ptycho.positions.size();
    for (int d = 0; d < num_batches; d++) {
      const unsigned int difpad_batch_zsize = ptycho.positions[d]->sizez;
      const size_t difpad_idx = d * ptycho.multibatchsize;

      cur_difpad.Resize(difpadshape.x, difpadshape.y, difpad_batch_zsize);
      cur_difpad.LoadToGPU(ptycho.cpudifpads + difpad_idx * difpadshape.x * difpadshape.y);

      for (int g = 0; g < ptycho.gpus.size(); g++) {
          const size_t difpadsizez = ptycho_cur_batch_gpu_zsize(ptycho, d, g);
        if (difpadsizez > 0) {
          SetDevice(ptycho.gpus, g);

          dim3 blk = ptycho.exitwave->ShapeBlock();
          blk.z = difpadsizez;
          dim3 thr = ptycho.exitwave->ShapeThread();

          Image<Position>& ptr_roi = *ptycho.positions[d]->arrays[g];

          KGLExitwave<<<blk, thr>>>(*ptycho.exitwave->arrays[g],
                  *ptycho.probe->arrays[g],
                  *ptycho.object->arrays[g], ptr_roi);

          project_reciprocal_space(ptycho, cur_difpad.arrays[g],
                  g, poscorr.isGradPm);

          KGLPs<<<blk, thr>>>(*ptycho.probe->arrays[g],
                  *ptycho.object_acc->arrays[g],
                  *ptycho.object_div->arrays[g],
                  *ptycho.exitwave->arrays[g], ptr_roi);
        }
      }
      if (bIterProbe) PosCorrectionProjectProbe(poscorr, d);
    }

    ssc_debug("Syncing OBJ and setting RF");
    if (ptycho.objmomentum >= 0)
        ptycho.object->WeightedLerpSync(
                *ptycho.object_acc, *ptycho.object_div,
                ptycho.objstep, ptycho.objmomentum,
                objvelocity, ptycho.objreg);

    if (ptycho.objectsupport != nullptr)
      for (int g = 0; g < ptycho.gpus.size(); g++) {
        SetDevice(ptycho.gpus, g);
        ApplySupport(*ptycho.object->arrays[g],
                *ptycho.objectsupport->arrays[g],
                ptycho.SupportSizes);
      }

    PosCorrectionApplyProbeUpdate(poscorr, probevelocity, ptycho.probestep, ptycho.probemomentum, ptycho.probereg);

    ptycho.cpurfact[iter] = sqrtf(ptycho.rfactors->SumCPU());

    if (iter % 10 == 0) {
        ssc_info(format("iter {}/{} error: {}",
                    iter, iterations, ptycho.cpurfact[iter]));
    }

  }

  auto time1 = ssc_time();
  ssc_info(format("End GL: {} ms", ssc_diff_time(time0, time1)));
}

