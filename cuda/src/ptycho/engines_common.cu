#include "hip/hip_runtime.h"
#include <cmath>
#include <common/types.hpp>
#include <common/logger.hpp>
#include <cmath>
#include <cstddef>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include "complex.hpp"
#include "engines_common.hpp"

//host position correction offsets
float const pos_offx[] = { 0, 1, -1, 0, 0, -1, -1, 1, 1};
float const pos_offy[] = { 0, 0, 0, 1, -1, -1, 1, -1, 1};
//device position correction offsets, should be the same as the host
__device__ float const d_pos_offx[] = { 0, 1, -1, 0, 0, -1, -1, 1, 1};
__device__ float const d_pos_offy[] = { 0, 0, 0, 1, -1, -1, 1, -1, 1};

// 4-neighborhood, 8-neighborhood
constexpr int n_pos_neighbors = 8;


__global__ void KSideExitwave(GArray<complex> exitwave, 
                              const GArray<complex> probe, 
                              const GArray<complex> object, 
                              const GArray<Position> positions, 
                              int offx, 
                              int offy)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= probe.shape.x)
        return;

    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z;

    if(true)
    {
        int objposx = idx + (int)positions(idz,0,0).x + offx;
        int objposy = idy + (int)positions(idz,0,0).y + offy;

        const complex& obj = object(objposy, objposx);

        for(size_t m=0; m<probe.shape.z; m++) // for each incoherent mode
            exitwave(m + probe.shape.z*blockIdx.z,idy,idx) = obj * probe(m,idy,idx);
    }
}
__global__ void KComputeError(float* error_errors_rfactor, 
                              const GArray<complex> exitwave, 
                              const GArray<float> diffraction_patterns, 
                              const float* background, 
                              size_t nummodes)
{
    __shared__ float shared_error_error_rfactor[64];

    if(threadIdx.x<64)
        shared_error_error_rfactor[threadIdx.x] = 0;

    __syncthreads();

    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t idy = blockIdx.y;

    // halo regions
    if(idx >= diffraction_patterns.shape.x)
        return;

    bool bApplyBkg = background != nullptr;

    float diff_pattern = diffraction_patterns(blockIdx.z, idy, idx);

    if(diff_pattern >= 0)
    {
        float wabs2 = 0.0f;
        if(bApplyBkg) wabs2 = sq(background[idy*diffraction_patterns.shape.x+idx]);

        for(int m=0; m<nummodes; m++)
            wabs2 += exitwave(blockIdx.z*nummodes + m, idy, idx).abs2();

        const int sigmask = (diff_pattern < 0);
        atomicAdd(shared_error_error_rfactor + threadIdx.x%64, sigmask * sq(sqrtf(diff_pattern)-sqrtf(wabs2)));
    }

    __syncthreads();

    Reduction::KSharedReduce(shared_error_error_rfactor,64);
    if(threadIdx.x==0)
        atomicAdd(error_errors_rfactor + blockIdx.z, shared_error_error_rfactor[0]);
}

__global__ void KProjectPhiToProbe(const GArray<complex> probe, complex* probe_acc, float* probe_div,
        const GArray<complex> object, const GArray<complex> exitwave, const GArray<Position> positions,
        bool bFTNorm, bool isGrad) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= probe.shape.x) return;

    complex pacc = complex(0);
    float pdiv = 0;

    for (size_t pos = 0; pos < positions.shape.z; pos++)
        for (int p = 0; p < positions.shape.x; p++)  // for each flyscan point
        {
            int objposx = idx + (int)positions(pos, 0, p).x;
            int objposy = idy + (int)positions(pos, 0, p).y;

            complex obj = object(objposy, objposx);
            complex ew = exitwave((pos * positions.shape.x + p) * probe.shape.z + blockIdx.z, idy, idx);

            pacc += ew * obj.conj();
            pdiv += obj.abs2();
        }

    size_t index = blockIdx.z * probe.shape.x * probe.shape.y + idy * probe.shape.x + idx;

    if (bFTNorm) pacc /= (float)(probe.shape.x * probe.shape.y);
    if (!isGrad) pacc -= probe[index] * pdiv;

    probe_acc[index] += pacc;
    probe_div[index] += pdiv;
}

// the kernel code is replicated for complex16, for some reason cuda was not playing well with explicit instantiation on gpu kernels
__global__ void KProjectPhiToProbe(const GArray<complex> probe, complex* probe_acc, float* probe_div,
        const GArray<complex> object, const GArray<complex16> exitwave, const GArray<Position> positions,
        bool bFTNorm, bool isGrad) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= probe.shape.x) return;

    complex pacc = complex(0);
    float pdiv = 0;

    for (size_t pos = 0; pos < positions.shape.z; pos++)
        for (int p = 0; p < positions.shape.x; p++)  // for each flyscan point
        {
            int objposx = idx + (int)positions(pos, 0, p).x;
            int objposy = idy + (int)positions(pos, 0, p).y;

            complex obj = object(objposy, objposx);
            complex ew = complex(exitwave((pos * positions.shape.x + p) * probe.shape.z + blockIdx.z, idy, idx));

            pacc += ew * obj.conj();
            pdiv += obj.abs2();
        }

    size_t index = blockIdx.z * probe.shape.x * probe.shape.y + idy * probe.shape.x + idx;

    if (bFTNorm) pacc /= (float)(probe.shape.x * probe.shape.y);
    if (!isGrad) pacc -= probe[index] * pdiv;

    probe_acc[index] += pacc;
    probe_div[index] += pdiv;
}

template <typename dtype>
void ProjectPhiToProbe(Ptycho& pt, int section, const MImage<dtype>& Phi, bool bNormalizeFFT, bool isGrad,
        hipStream_t stream) {
    dim3 blk = pt.probe->ShapeBlock();
    dim3 thr = pt.probe->ShapeThread();

    for (int g = 0; g < pt.gpus.size(); g++) {
        SetDevice(pt.gpus, g);

        KProjectPhiToProbe<<<blk, thr, 0, stream>>>(
                pt.probe->arrays[g][0], pt.probe_num->Ptr(g), pt.probe_div->Ptr(g),
                pt.object->arrays[g][0], Phi.arrays[g][0], pt.positions[section]->arrays[g][0],
                bNormalizeFFT, isGrad);
    }
}

template void ProjectPhiToProbe<complex>(Ptycho& pt, int section,
        const cMImage& Phi, bool bNormalizeFFT, bool isGrad, hipStream_t st);

template void ProjectPhiToProbe<complex16>(Ptycho& pt, int section,
        const hcMImage& Phi, bool bNormalizeFFT, bool isGrad, hipStream_t st);

extern "C" {
    void EnablePeerToPeer(const std::vector<int>& gpus);
    void DisablePeerToPeer(const std::vector<int>& gpus);

    __global__ void KProjectReciprocalSpace(GArray<complex> exitwave,  
                                            const GArray<float> diffraction_patterns, 
                                            float* error_error_rfactor, 
                                            size_t upsample, 
                                            size_t nummodes,  
                                            bool isGrad) {

        __shared__ float shared_error_error_rfactor[64];

        if (threadIdx.x < 64) shared_error_error_rfactor[threadIdx.x] = 0;

        __syncthreads();

        const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
        const size_t idy = blockIdx.y;
        const size_t idz = blockIdx.z;

        if (idx >= diffraction_patterns.shape.x) return;

        const float diff_pattern = diffraction_patterns(idz, idy, idx);
        const float sqrt_difpad = sqrtf(diff_pattern);

        float exit_wave_factor = 1.0f;
        float exit_wave_addend = 0.0f;

        if (diff_pattern >= 0) {
            float wabs2 = 0.0f;
            float wabs = 0.0f;

            for (int m = 0; m < nummodes; m++)
                for (int v = 0; v < upsample; v++)
                    for (int u = 0; u < upsample; u++)
                        wabs2 += exitwave(idz * nummodes + m,  v + idy * upsample,  u + idx * upsample).abs2();

            wabs = sqrtf(wabs2) / upsample; // can we kill upsample? not sure it is necessary anymore.

            atomicAdd(shared_error_error_rfactor + threadIdx.x % 64, sq(sqrt_difpad - wabs));

            // Define ew_f and ew_a to be used in the next loop in ew = ew_f * ew + ew_a
            if (wabs > 0.0f && isGrad) {  //AP
                exit_wave_factor = (sqrt_difpad / wabs - 1); // why -1 for AP and not for RAAR?
                exit_wave_addend = 0.0f;
            }
            else if (wabs > 0.0f && !isGrad) { //RAAR
                exit_wave_factor = sqrt_difpad / wabs; // why -1 for AP and not for RAAR?
                exit_wave_addend = 0.0f;
            } else { // wabs <= 0.0f
                exit_wave_addend = sqrt_difpad;
                exit_wave_factor = 0.0f;
            }

        } else if (isGrad) { // if diff_pattern < 0 and isGrad. Make invalid points to be zero in the wavefront
            exit_wave_factor = 0.0f;
            exit_wave_addend = 0.0f;
        }

        for (int m = 0; m < nummodes; m++)
            for (int v = 0; v < upsample; v++)
                for (int u = 0; u < upsample; u++) {
                    complex ew = exitwave(idz * nummodes + m,  v + idy * upsample,   u + idx * upsample);

                    // application of the measured intensity to the exitwave (projection in reciprocal space)
                    ew = ew * exit_wave_factor + exit_wave_addend; 
                    exitwave(idz * nummodes + m,  v + idy * upsample,   u + idx * upsample) = ew;
                }

        __syncthreads();

        Reduction::KSharedReduce(shared_error_error_rfactor, 64);
        if (threadIdx.x == 0) atomicAdd(error_error_rfactor + blockIdx.y, shared_error_error_rfactor[0]);
    }
}


void ProjectReciprocalSpace(Ptycho &pt, rImage* diff_pattern, cImage* wavefront, int g, bool isGrad, hipStream_t stream) {

    SetDevice(pt.gpus, g);

    complex* ewave = wavefront->gpuptr;

    int upsample = wavefront->sizex / diff_pattern->sizex;

    if (upsample>1){
        printf("Upsample factor >1: ", upsample);
    }

    pt.propagator[g]->Propagate(ewave, ewave, wavefront->Shape(), 1, stream);

    wavefront->FFTShift2(stream);

    KProjectReciprocalSpace<<<diff_pattern->ShapeBlock(), diff_pattern->ShapeThread(), 0, stream>>>(*wavefront, 
                                                                                                    *diff_pattern, 
                                                                                                    pt.error->Ptr(g), 
                                                                                                    upsample,  
                                                                                                    pt.probe->sizez, 
                                                                                                    isGrad);

    wavefront->FFTShift2(stream);

    pt.propagator[g]->Propagate(ewave, ewave, wavefront->Shape(), -1, stream);

}


void ProjectReciprocalSpace(Ptycho &pt, rImage* diff_pattern, int g, bool isGrad, hipStream_t stream) {

    SetDevice(pt.gpus, g);

    complex* ewave = pt.wavefront->Ptr(g);

    int upsample = pt.wavefront->sizex / diff_pattern->sizex;

    pt.propagator[g]->Propagate(ewave, ewave, pt.wavefront->Shape(), 1, stream);

    pt.wavefront->arrays[g]->FFTShift2(stream);

    KProjectReciprocalSpace<<<diff_pattern->ShapeBlock(), diff_pattern->ShapeThread(), 0, stream>>>(pt.wavefront->arrays[g][0], 
                                                                                                    *diff_pattern, 
                                                                                                    pt.error->Ptr(g), 
                                                                                                    upsample, 
                                                                                                    pt.probe->sizez, 
                                                                                                    isGrad);

    pt.wavefront->arrays[g]->FFTShift2(stream);

    pt.propagator[g]->Propagate(ewave, ewave, pt.wavefront->Shape(), -1, stream);

}

void ApplyProbeSupport(Ptycho& pt) {
    SetDevice(pt.gpus, 0);
    const dim3 shape = dim3(pt.probe->sizex, pt.probe->sizey, pt.probe->sizez);
    complex *probe_ptr = pt.probe->arrays[0]->gpuptr;
    if (pt.distance_m > 0)
        pt.probepropagator->Propagate(probe_ptr, probe_ptr, shape, +pt.distance_m);

    pt.probe->arrays[0][0] *= pt.probesupport->arrays[0][0];

    if (pt.distance_m > 0)
        pt.probepropagator->Propagate(probe_ptr, probe_ptr, shape, -pt.distance_m);

    pt.probe->BroadcastSync();
}


void ApplyProbeUpdate(Ptycho& pt, cImage& velocity, float stepsize, float momentum, float epsilon) {

    if (momentum < 0 | stepsize < 0) return;

    SetDevice(pt.gpus, 0);

    pt.probe->WeightedLerpSync(*(pt.probe_num), *(pt.probe_div), stepsize, momentum, velocity, epsilon);

    if (pt.probesupport != nullptr) {
        ApplyProbeSupport(pt);
    }
}

__global__
void KPositionCorrection(float* errorcounter, Position* positions,
        const size_t batchsize,
        const dim3 objshape, const dim3 probeshape) {

    const int z = blockIdx.x*blockDim.x + threadIdx.x;

    if (z >= batchsize)
        return;

    float* error = errorcounter + z;
    float minerror = error[0];
    int minidx = 0;

    for(int k = 1; k <= n_pos_neighbors; k++) {
        if(error[batchsize*k] < minerror) {
            minerror = error[batchsize*k];
            minidx = k;
        }
    }

    const float x = positions[z].x;
    const float y = positions[z].y;
    positions[z].x = fminf(fmaxf(x+d_pos_offx[minidx],1.1f),
            objshape.x - probeshape.x-3);
    positions[z].y = fminf(fmaxf(y+d_pos_offy[minidx],1.1f),
            objshape.y - probeshape.y-3);

}

void ApplyPositionCorrection(Ptycho& ptycho) {

    ptycho.errorcounter->SetGPUToZero();

    const dim3 difpadshape = ptycho.diff_pattern_shape;
    rMImage cur_difpad(difpadshape.x, difpadshape.y, ptycho.multibatchsize,
            false, ptycho.gpus, MemoryType::EAllocGPU);

    const size_t batchsize = ptycho.positions[0]->arrays[0]->sizez;
    const size_t num_batches = PtychoNumBatches(ptycho);
    const size_t ngpus = PtychoNumGpus(ptycho);
    for(int d = 0; d < num_batches; d++) {
        const size_t difpad_batch_zsize = PtychoCurBatchZsize(ptycho, d);
        const size_t difpad_idx = d * PtychoBatchSize(ptycho);

        cur_difpad.Resize(difpadshape.x, difpadshape.y, difpad_batch_zsize);
        cur_difpad.LoadToGPU(ptycho.cpu_diff_pattern + difpad_idx * difpadshape.x * difpadshape.y);
        for(int g = 0; g < ngpus; g++) {
            for(int k = 0; k <= n_pos_neighbors; k++) {
                SetDevice(ptycho.gpus, g);
                const size_t difpadsizez = ptycho.positions[d][0][g].sizez;
                if(difpadsizez > 0) {
                    dim3 blk = ptycho.wavefront->ShapeBlock();
                    blk.z = difpadsizez;
                    dim3 thr = ptycho.wavefront->ShapeThread();

                    Image<Position>& ptr_roi = *ptycho.positions[d]->arrays[g];
                    KSideExitwave<<<blk,thr>>>(*ptycho.wavefront->arrays[g],  *ptycho.probe->arrays[g], *ptycho.object->arrays[g], ptr_roi, pos_offx[k], pos_offy[k]);

                    ptycho.propagator[g]->Propagate(ptycho.wavefront->arrays[g]->gpuptr,  ptycho.wavefront->arrays[g]->gpuptr,  ptycho.wavefront->arrays[g]->Shape(), 1);

                    // compute errors 
                    KComputeError<<<blk,thr>>>(ptycho.errorcounter->arrays[g]->gpuptr + batchsize*k,
                                               *ptycho.wavefront->arrays[g], 
                                               *cur_difpad.arrays[g],
                                               nullptr,
                                               ptycho.probe->sizez);
                }
            }
        }

        for(int g = 0; g < ngpus; g++) {
            SetDevice(ptycho.gpus, g);
            const size_t batch_size = PtychoCurBatchGpuZsize(ptycho, d, g);
            if (batch_size > 0) {
                KPositionCorrection<<<256, batchsize / 256 + (batchsize % 256 > 0)>>>
                    (ptycho.errorcounter->arrays[g]->gpuptr,
                     ptycho.positions[d][0][g].gpuptr, batchsize,
                     ptycho.object->Shape(), ptycho.probe->Shape());
            }
        }
    }
}

void DestroyPtycho(Ptycho*& ptycho_ref) {
    Ptycho& ptycho = *ptycho_ref;
    sscDebug("Dealloc POpt.");
    if (ptycho.object_div) delete ptycho.object_div;
    ptycho.object_div = nullptr;
    if (ptycho.object_num) delete ptycho.object_num;
    ptycho.object_num = nullptr;
    if (ptycho.probe_div) delete ptycho.probe_div;
    ptycho.probe_div = nullptr;
    if (ptycho.probe_num) delete ptycho.probe_num;
    ptycho.probe_num = nullptr;

    //hipHostFree(ptycho.cpu_diff_pattern);
    hipHostUnregister(ptycho.cpu_diff_pattern);

    sscDebug("Deallocating base algorithm.");
    for (int g = 0; g < ptycho.gpus.size(); g++) {
        sscDebug(format("Dealloc propagator: {}", g));
        SetDevice(ptycho.gpus, g);
        delete ptycho.propagator[g];
        ptycho.propagator[g] = nullptr;
    }

    sscDebug("Probe D2H");
    ptycho.probe->CopyTo(ptycho.cpuprobe);
    sscDebug("Object D2H");
    ptycho.object->CopyTo(ptycho.cpuobject);

    sscDebug("Dealloc probe.");
    delete ptycho.probe;
    sscDebug("Dealloc object.");
    delete ptycho.object;
    sscDebug("Dealloc exitwave.");
    delete ptycho.wavefront;

    sscDebug("Dealloc supports.");
    if (ptycho.objectsupport != nullptr) delete ptycho.objectsupport;
    if (ptycho.probesupport != nullptr) delete ptycho.probesupport;

    sscDebug("Dealloc error_errors_rfactor.");
    delete ptycho.error;

    sscDebug("Dealloc errorcounter.");
    delete ptycho.errorcounter;

    sscDebug("Dealloc rois.");
    for (auto* pos : ptycho.positions) delete pos;

    sscDebug("Done.");

    SetDevice(ptycho.gpus, 0);
    delete ptycho.probepropagator;

    ptycho_ref = nullptr;
}

Ptycho* CreatePtycho(float* _difpads, const dim3& difshape, complex* _probe, const dim3& probeshape,
        complex* _object, const dim3& objshape, Position* positions, int numrois, int batchsize,
        float* _rfact, const std::vector<int>& gpus, float* _objectsupport, float* _probesupport,
        int numobjsupp,  float wavelength_m, float pixelsize_m, float distance_m,
        int poscorr_iter,
        float step_obj, float step_probe,
        float reg_obj, float reg_probe) {

    Ptycho* ptycho = new Ptycho;
    ptycho->gpus = gpus;

    sscDebug("Initializing algorithm.");
    sscDebug("Enabling P2P");

    ptycho->pixelsize_m = pixelsize_m;
    ptycho->wavelength_m = wavelength_m;
    ptycho->distance_m = distance_m;

    EnablePeerToPeer(ptycho->gpus);

    ptycho->objreg = reg_obj;
    ptycho->probereg = reg_probe;
    ptycho->objstep = step_obj;
    ptycho->probestep = step_probe;

    ptycho->diff_pattern_shape.x = difshape.x;
    ptycho->diff_pattern_shape.y = difshape.y;
    ptycho->diff_pattern_shape.z = difshape.z;

    ptycho->poscorr_iter = poscorr_iter;

    const int ngpus = gpus.size();
    if (batchsize > 0) {
        ptycho->singlebatchsize = batchsize;
        ptycho->multibatchsize = batchsize * ngpus;

        batchsize *= ngpus;
    } else {
        ptycho->singlebatchsize = (numrois + ngpus - 1) / ngpus;
        batchsize = ptycho->multibatchsize = ptycho->singlebatchsize * ngpus;
    }
    sscDebug(format("Batches: {} {}", ptycho->singlebatchsize, ptycho->multibatchsize));

    ptycho->total_num_rois = numrois;

    ptycho->cpu_diff_pattern = _difpads;

    size_t difpad_size = ptycho->diff_pattern_shape.x * ptycho->diff_pattern_shape.y * ptycho->diff_pattern_shape.z;
    //hipHostMalloc(&(ptycho->cpu_diff_pattern), difpad_size * sizeof(float));
    //hipMemcpy(ptycho->cpu_diff_pattern, _difpads, difpad_size * sizeof(float), hipMemcpyHostToHost);

    ptycho->cpu_diff_pattern = _difpads;
    hipHostRegister(ptycho->cpu_diff_pattern, difpad_size * sizeof(float), hipHostRegisterDefault);

    ptycho->cpuprobe = _probe;
    ptycho->cpuobject = _object;
    ptycho->cpupositions = positions;
    ptycho->cpuerror = _rfact;

    sscDebug("Alloc probe.");
    ptycho->probe = new cMImage(_probe, probeshape, true, gpus);
    sscDebug("Alloc obj");
    ptycho->object = new cMImage(_object, objshape, true, gpus);
    sscDebug("Alloc EW");
    ptycho->wavefront = new cMImage(probeshape.x, probeshape.y,
            ptycho->singlebatchsize * probeshape.z, true, gpus);

    sscDebug("Alloc Supports");

    if (numobjsupp > 0 && _objectsupport != nullptr) {
        ptycho->objectsupport = new rMImage(_objectsupport, dim3(objshape.x, objshape.y, numobjsupp), true, ptycho->gpus);
        ptycho->SupportSizes = std::vector<float>();
        for (int i = 0; i < numobjsupp; i++) {
            float s = 0;
            for (int j = 0; j < objshape.x * objshape.y; j++) s += _objectsupport[j + i * objshape.x * objshape.y];
            ptycho->SupportSizes.push_back(s);
        }
    } else
        ptycho->objectsupport = nullptr;

    if (_probesupport != nullptr)
        ptycho->probesupport = new rMImage(_probesupport, dim3(probeshape.x, probeshape.y, probeshape.z), true, ptycho->gpus);
    else
        ptycho->probesupport = nullptr;

    sscDebug("Alloc r-factor error");
    ptycho->error = new rMImage(difshape.y, 1, 1, true, ptycho->gpus);
    ptycho->error->SetGPUToZero();

    ptycho->errorcounter = new rMImage(n_pos_neighbors + 1, 1, batchsize, true, ptycho->gpus);

    SetDevice(ptycho->gpus, 0);
    ptycho->roibatch_offset = std::vector<int>();

    for (size_t n = 0; n < numrois; n += batchsize) {
        sscDebug(format("Creating DPGroup at: {} of {} at step {}" , n, numrois, batchsize));
        if (numrois - n < batchsize)  // last batch
        {
            ptycho->positions.push_back(new PositionArray(positions + n, 1, 1, numrois - n, false, gpus));
        } else {
            ptycho->positions.push_back(new PositionArray(positions + n, 1, 1, batchsize, false, gpus));
        }
        ptycho->roibatch_offset.push_back(n / ngpus);
    }

    for (int g = 0; g < gpus.size(); g++) {
        SetDevice(gpus, g);
        sscDebug(format("Creating propagator: {}", g));
        ptycho->propagator[g] = new Fraunhoffer();
    }

    sscDebug("Computing I0");
    SetDevice(gpus, 0);
    ptycho->probepropagator = new ASM(wavelength_m, pixelsize_m);

    ptycho->object_div = new rMImage(ptycho->object->Shape(), true, gpus);
    ptycho->object_num = new cMImage(ptycho->object->Shape(), true, gpus);

    ptycho->probe_div = new rMImage(ptycho->probe->Shape(), true, gpus);
    ptycho->probe_num = new cMImage(ptycho->probe->Shape(), true, gpus);

    return ptycho;
}
